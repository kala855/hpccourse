#include "hip/hip_runtime.h"
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <time.h>
#include <hip/hip_runtime.h>

#define RED 2
#define GREEN 1
#define BLUE 0

using namespace cv;


__global__ void img2gray(unsigned char *imageInput, int width, int height, unsigned char *imageOutput){
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if((row < height) && (col < width)){
        imageOutput[row*width+col] = imageInput[(row*width+col)*3+RED]*0.299 + imageInput[(row*width+col)*3+GREEN]*0.587 \
                                     + imageInput[(row*width+col)*3+BLUE]*0.114;
    }
}


int main(int argc, char **argv){
    hipError_t error = hipSuccess;
    clock_t start, end, startGPU, endGPU;
    double cpu_time_used, gpu_time_used;
    char* imageName = argv[1];
    unsigned char *dataRawImage, *d_dataRawImage, *d_imageOutput, *h_imageOutput;
    Mat image;
    image = imread(imageName, 1);

    if(argc !=2 || !image.data){
        printf("No image Data \n");
        return -1;
    }

    Size s = image.size();

    int width = s.width;
    int height = s.height;
    int size = sizeof(unsigned char)*width*height*image.channels();
    int sizeGray = sizeof(unsigned char)*width*height;


    dataRawImage = (unsigned char*)malloc(size);
    error = hipMalloc((void**)&d_dataRawImage,size);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_dataRawImage\n");
        exit(-1);
    }

    h_imageOutput = (unsigned char *)malloc(sizeGray);
    error = hipMalloc((void**)&d_imageOutput,sizeGray);
    if(error != hipSuccess){
        printf("Error reservando memoria para d_imageOutput\n");
        exit(-1);
    }


    dataRawImage = image.data;

    startGPU = clock();
    error = hipMemcpy(d_dataRawImage,dataRawImage,size, hipMemcpyHostToDevice);
    if(error != hipSuccess){
        printf("Error copiando los datos de dataRawImage a d_dataRawImage \n");
        exit(-1);
    }

    int blockSize = 32;
    dim3 dimBlock(blockSize,blockSize,1);
    dim3 dimGrid(ceil(width/float(blockSize)),ceil(height/float(blockSize)),1);
    img2gray<<<dimGrid,dimBlock>>>(d_dataRawImage,width,height,d_imageOutput);
    hipDeviceSynchronize();
    hipMemcpy(h_imageOutput,d_imageOutput,sizeGray,hipMemcpyDeviceToHost);
    endGPU = clock();

    Mat gray_image;
    gray_image.create(height,width,CV_8UC1);
    gray_image.data = h_imageOutput;

    start = clock();
    Mat gray_image_opencv;
    cvtColor(image, gray_image_opencv, COLOR_RGB2GRAY );
    end = clock();


    imwrite("./Gray_Image.jpg",gray_image);

    //free(dataRawImage);
    gpu_time_used = ((double) (endGPU - startGPU)) / CLOCKS_PER_SEC;
    printf("Tiempo Algoritmo Paralelo: %.10f\n",gpu_time_used);
    cpu_time_used = ((double) (end - start)) /CLOCKS_PER_SEC;
    printf("Tiempo Algoritmo OpenCV: %.10f\n",cpu_time_used);
    printf("La aceleración obtenida es de %.10fX\n",cpu_time_used/gpu_time_used);

    hipFree(d_dataRawImage);
    hipFree(d_imageOutput);
    return 0;
}
